#include "hip/hip_runtime.h"
// MP 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
	//@@ Insert code to implement vector addition here
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<len)
	{
		//printf("in1[%d]: %d\n", i, in1[i]);
		//printf("in2[%d]: %d\n", i, in2[i]);
		out[i] = in1[i] + in2[i];
		//printf("out[%d]: %d\n", i, out[i]);
	}
}


int main(int argc, char **argv) {
	wbArg_t args;
	int inputLength;
	float *hostInput1;
	float *hostInput2;
	float *hostOutput;
	float *deviceInput1;
	float *deviceInput2;
	float *deviceOutput;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
	hostOutput = (float *)malloc(inputLength * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
	//@@ Allocate GPU memory here

	int size = inputLength * sizeof(float);
	wbLog(TRACE, "size is ", size);
	hipError_t err0 = hipMalloc((void **)&deviceInput1, size); //address of pointer, size
	if (err0 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err0), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	hipError_t err1 = hipMalloc((void **)&deviceInput2, size);
	if (err1 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipError_t err2 = hipMalloc((void **)&deviceOutput, size);
	if (err2 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	//@@ Copy memory to the GPU here
	hipError_t err3 = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice); //dest, src, size, hipMemcpyHostToDevice
	if (err3 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	hipError_t err4 = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (err4 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err4), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	//@@ Initialize the grid and block dimensions here
	//dim3 DimGrid(ceil(size/256.0),1,1);
	//dim3 DimBlock(256,1,1);
	int numBlocks = ceil(size / 256.0);
	int threadsPerBlock = 256;
	wbLog(TRACE, "There are ", numBlocks, " blocks");
	wbLog(TRACE, "There are ", threadsPerBlock, " threads per block");

	wbTime_start(Compute, "Performing CUDA computation");
	//@@ Launch the GPU Kernel here
	//vecAdd<<<DimGrid,DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, size);
	vecAdd << <numBlocks, threadsPerBlock >> >(deviceInput1, deviceInput2, deviceOutput, size);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	//@@ Copy the GPU memory back to the CPU here
	hipError_t err5 = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost); //dest, src, size, hipMemcpyDeviceToHost
	if (err5 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err5), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	//@@ Free the GPU memory here
	hipError_t err6 = hipFree(deviceInput1);
	if (err6 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err6), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	hipError_t err7 = hipFree(deviceInput2);
	if (err7 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err7), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	hipError_t err8 = hipFree(deviceOutput);
	if (err8 != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err8), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, inputLength);

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

	return 0;
}
